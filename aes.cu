#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include "aes.h"

__constant__ unsigned char D_SBOX[16][16];
__constant__ unsigned char D_INV_SBOX[16][16];

__device__ void keyExpansion() {

}

__global__ void AESKernel(char *x)
{
	printf("%02x %d %d\n", D_INV_SBOX[threadIdx.x][threadIdx.y], threadIdx.x, threadIdx.y);
	//printf("%02x", D_SBOX[threadIdx.x]);
}

int main(int argc, char **argv)
{
	AES_INFO *data = get_args(argv, argc);
	char *h = (char *)malloc(256);
	for (int i = 0; i < 256; i++)
		h[i] = i;

	char *arr;
	hipMalloc((void**)&arr, sizeof(char) * 256);
	hipMemset(arr, 0, sizeof(char)*256);

	if (hipMemcpyToSymbol(HIP_SYMBOL(D_SBOX), SBOX, sizeof(char) * 256) != hipSuccess) {
		printf("error on copy");
	}
	if (hipMemcpyToSymbol(HIP_SYMBOL(D_INV_SBOX), INVERSE_SBOX, sizeof(char) * 256) != hipSuccess) {
		printf("error on copy");
	}

	hipMemcpy(arr, h, 256, hipMemcpyHostToDevice);

	dim3 block(16, 16);

	AESKernel << <1, block>> > (arr);
	hipError_t err = hipGetLastError();
	if(err != hipSuccess)
		fprintf(stderr, "err %s", hipGetLastError());

	hipMemcpy(h, arr, 256, hipMemcpyDeviceToHost);
    return 0;
}

//arg[0] - program name - not used
//arg[1] - Encrypt or decrypt(e for encrypt, d for decrypt)
//arg[2] - Filename containing our key
//arg[3] - Input file to encrypt / decrypt
//arg[4] - Output file name
AES_INFO *get_args(char **argv, int argc) {
	AES_INFO *info = (AES_INFO *)malloc(sizeof(AES_INFO));
	if (NULL == info) {
		printf("Error on malloc.\n");
		exit(EXIT_FAILURE);
	}

	if (argc != 5) {
		printf("Incorrect number of arguments. Found %d\n", argc);
		exit(EXIT_FAILURE);
	}

	switch (*argv[1]) {
	case 'e':
		info->mode = ENCRYPT;
		break;
	case 'd':
		info->mode = DECRYPT;
		break;
	default:
		printf("ERROR. First argument should be e for Encrypt or d for Decrypt.\n");
		exit(EXIT_FAILURE);
	}

	//get the key
	char buffer[KEYSIZE];

	FILE *key_fp = fopen(argv[2], "rb");
	if (NULL == key_fp) {
		printf("Error opening keyfile: %s\n", argv[2]);
		exit(EXIT_FAILURE);
	}

	fread(buffer, 1, KEYSIZE, key_fp);
	strncpy(info->key, buffer, KEYSIZE);
	
	printf("Using key: ");
	for (int i = 0; i < KEYSIZE; i++) {
		printf("%02x ", info->key[i]);
	}

	//get FILE * to file to encrypt/decrypt
	info->fin = fopen(argv[3], "rb");
	if (NULL == info->fin) {
		printf("Error opening input file %s\n", argv[3]);
		exit(EXIT_FAILURE);
	}
	printf("\nUsing input file %s\n", argv[3]);

	//get output file name to be used later
	info->output_filename = (char *)malloc(strlen(argv[4]) + 1);
	strcpy(info->output_filename, argv[4]);
	printf("Output file is %s\n", argv[4]);

	return info;
}
